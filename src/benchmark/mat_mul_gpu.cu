#include "hip/hip_runtime.h"
/**
 * @file mat_mul_gpu.cu
 * @author your name (you@domain.com)
 * @brief 
 * @version 0.1
 * @date 2023-01-13
 * 
 * @copyright Copyright (c) 2023
 * 
 */

#include "benchmark/mat_mul_gpu.cuh"

#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>
#include "parco/Timer.cuh"
#include "parco/CheckError.cuh"


const int BLOCK_SIZE_X = 16;
const int BLOCK_SIZE_Y = 16;

__global__
void matrixMultiplicationKernel(const float* d_matrixA,
                                const float* d_matrixB,
                                std::size_t        N,
                                float*       d_matrixC) {
    // Calculate the row index of the Pd element and M
    std::size_t Row = blockIdx.y*blockDim.y + threadIdx.y;
    // Calculate the column index of Pd and N
    std::size_t Col = blockIdx.x*blockDim.x + threadIdx.x;
    
    float Pvalue = 0;
    // each thread computes one element of the block sub-matrix
    for (std::size_t k = 0; k < N; ++k)
        Pvalue += d_matrixA[Row*N+k] * d_matrixB[Col+k*N];
    
    d_matrixC[Row*N+Col] = Pvalue;
}

CudaMem mat_mul_gpu_init(const std::size_t N)
{
    CudaMem info;
    SAFE_CALL( hipMalloc( &info.d_matrixA, N*N * sizeof(float) ));
    SAFE_CALL( hipMalloc( &info.d_matrixB, N*N * sizeof(float) ));
    SAFE_CALL( hipMalloc( &info.d_matrixC, N*N * sizeof(float) ));
    return info;
}


void mat_mul_gpu_deinit(CudaMem info)
{
    SAFE_CALL( hipFree( info.d_matrixA ) );
    SAFE_CALL( hipFree( info.d_matrixB ) );
    SAFE_CALL( hipFree( info.d_matrixC ) );
}


float* mat_mul_gpu(float* dst, const float* src1, const float* src2, const std::size_t N)
{
    float *d_matrixA, *d_matrixB, *d_matrixC;
    SAFE_CALL( hipMalloc( &d_matrixA, N*N * sizeof(float) ));
    SAFE_CALL( hipMalloc( &d_matrixB, N*N * sizeof(float) ));
    SAFE_CALL( hipMalloc( &d_matrixC, N*N * sizeof(float) ));

    // -------------------------------------------------------------------------
    // COPY DATA FROM HOST TO DEVIE
    SAFE_CALL( hipMemcpy( d_matrixA, src1, N*N * sizeof(float), hipMemcpyHostToDevice));
    SAFE_CALL( hipMemcpy( d_matrixB, src2, N*N * sizeof(float), hipMemcpyHostToDevice));

    // -------------------------------------------------------------------------
    // DEVICE INIT
    dim3 DimGrid(N/BLOCK_SIZE_X, N/BLOCK_SIZE_Y, 1);
    if (N%BLOCK_SIZE_X) DimGrid.x++;
    if (N%BLOCK_SIZE_Y) DimGrid.y++;
    dim3 DimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
    
    // -------------------------------------------------------------------------
    // DEVICE EXECUTION
    matrixMultiplicationKernel<<< DimGrid,DimBlock>>> (d_matrixA, d_matrixB, N, d_matrixC);
    CHECK_CUDA_ERROR

    // -------------------------------------------------------------------------
    // COPY DATA FROM DEVICE TO HOST
    SAFE_CALL( hipMemcpy( dst, d_matrixC, N*N * sizeof(float), hipMemcpyDeviceToHost));

    SAFE_CALL( hipFree( d_matrixA ) );
    SAFE_CALL( hipFree( d_matrixB ) );
    SAFE_CALL( hipFree( d_matrixC ) );

    return dst;
}


float* mat_mul_gpu_exec(float* dst, const float* src1, const float* src2, const std::size_t N, CudaMem info) 
{
    // -------------------------------------------------------------------------
    // COPY DATA FROM HOST TO DEVIE
    SAFE_CALL( hipMemcpy( info.d_matrixA, src1, N*N * sizeof(float), hipMemcpyHostToDevice));
    SAFE_CALL( hipMemcpy( info.d_matrixB, src2, N*N * sizeof(float), hipMemcpyHostToDevice));

    // -------------------------------------------------------------------------
    // DEVICE INIT
    dim3 DimGrid(N/BLOCK_SIZE_X, N/BLOCK_SIZE_Y, 1);
    if (N%BLOCK_SIZE_X) DimGrid.x++;
    if (N%BLOCK_SIZE_Y) DimGrid.y++;
    dim3 DimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
    
    // -------------------------------------------------------------------------
    // DEVICE EXECUTION
    matrixMultiplicationKernel<<< DimGrid,DimBlock>>> (info.d_matrixA, info.d_matrixB, N, info.d_matrixC);
    CHECK_CUDA_ERROR

    // -------------------------------------------------------------------------
    // COPY DATA FROM DEVICE TO HOST
    SAFE_CALL( hipMemcpy( dst, info.d_matrixC, N*N * sizeof(float), hipMemcpyDeviceToHost));

    return dst;
}